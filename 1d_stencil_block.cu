// UCSC CMPE220 Advanced Parallel Processing
// Prof. Heiner Leitz
// Author: Marcelo Siero.
// Modified from code by:: Andreas Goetz (agoetz@sdsc.edu)
// CUDA program to perform 1D stencil operation in parallel on the GPU
//
// /* FIXME */ COMMENTS ThAT REQUIRE ATTENTION


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <cuda_device_runtime_api.h>
#include <chrono>

// define vector length, stencil radius,
#define INPUTSIZE 64l
#define N (1024*1024*INPUTSIZE)
#define RADIUS 3
#define GRIDSIZE 128
#define BLOCKSIZE 256

int gridSize  = GRIDSIZE;
int blockSize = BLOCKSIZE;

float milliseconds = 0;

void cudaErrorCheck()
{
  hipError_t error = hipGetLastError();
  const char* errorName = hipGetErrorName(error);
  std::string s0(errorName);
  std::cout << ">>>> " <<  s0 << std::endl;

  const char* errorDescription = hipGetErrorString(error);
  std::string s1(errorDescription);
  std::cout << ">>>> " << s1 << std::endl;
}

void start_timer(hipEvent_t* start) {
   // FIXME: ADD TIMING CODE, HERE, USE GLOBAL VARIABLES AS NEEDED.
   hipEventCreate(start);
   hipEventRecord(*start);
}

float stop_timer(hipEvent_t* start, hipEvent_t* stop) {
   // FIXME: ADD TIMING CODE, HERE, USE GLOBAL VARIABLES AS NEEDED.
   hipEventCreate(stop);
   hipEventRecord(*stop);

   hipEventSynchronize(*stop);
   hipEventElapsedTime(&milliseconds, *start, *stop);
   return(milliseconds);
}

hipDeviceProp_t prop;
int device;
void getDeviceProperties() {
   // FIXME: Implement this function so as to acquire and print the following
   // device properties:
   //    Major and minor CUDA capability, total device global memory,
   //    size of shared memory per block, number of registers per block,
   //    warp size, max number of threads per block, number of multi-prccessors
   //    (SMs) per device, Maximum number of threads per block dimension (x,y,z),
   //    Maximumum number of blocks per grid dimension (x,y,z).
   //
   // These properties can be useful to dynamically optimize programs.  For
   // instance the number of SMs can be useful as a heuristic to determine
   // how many is a good number of blocks to use.  The total device global
   // memory might be important to know just how much data to operate on at
   // once.

   hipGetDevice(&device);
   hipGetDeviceProperties(&prop, device);

   std::cout << "The major and minor cuda capabilities are: \t\t" << prop.major << ", "<< prop.minor << std::endl;
   std::cout << "The total device global memory is : \t\t" << static_cast<int>(prop.totalGlobalMem) << " bytes" << std::endl;
   std::cout << "The size of shared memory per block is : \t\t" << static_cast<int>(prop.sharedMemPerBlock) << " bytes" << std::endl;
   std::cout << "The number of registers per block is: \t\t" << prop.regsPerBlock << std::endl;
   std::cout << "The warp size is : \t\t" << prop.warpSize << " threads" << std::endl;
   std::cout << "The max number of threads per block is : \t\t" << prop.maxThreadsPerBlock << std::endl;
   std::cout << "The number of multiprocessors is : \t\t" << prop.multiProcessorCount << " per device" << std::endl;
   std::cout << "The number of Maximum number of threads per block dimension (x,y,z) per device: \t\t" << prop.maxThreadsDim[0] <<", "<< prop.maxThreadsDim[1] << ", "<< prop.maxThreadsDim[2] << std::endl;
   std::cout << "The Maximumum number of blocks per grid dimension " << prop.maxGridSize[0] << ", " << prop.maxGridSize[0] <<", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << std::endl;

}

void newline() { std::cout << std::endl; };

void printThreadSizes() {
   int noOfThreads = gridSize * blockSize;
   printf("Blocks            = %d\n", gridSize);  // no. of blocks to launch.
   printf("Threads per block = %d\n", blockSize); // no. of threads to launch.
   printf("Total threads     = %d\n", noOfThreads);
   printf("Number of grids   = %d\n", (N + noOfThreads -1)/ noOfThreads);
}

// -------------------------------------------------------
// CUDA device function that performs 1D stencil operation
// -------------------------------------------------------
__global__ void stencil_1D_block(int *in, int *out, long dim){

  long gindex = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = gridDim.x * blockDim.x;

  __shared__ int temp[BLOCKSIZE + 2*RADIUS];

  int lindex = threadIdx.x + RADIUS;
  /* FIXME PART 2 - MODIFIY PROGRAM TO USE SHARED MEMORY. */

    //read all gindex elements into the temp array
  temp[lindex] = in[gindex];

  if (gindex < RADIUS)   //for the first 3 threads in the grid
  {
      temp[lindex - RADIUS] = 0;
      temp[lindex + BLOCKSIZE] = in[gindex + BLOCKSIZE];

  }

  else if (gindex >= (stride - RADIUS)) //last three threads in the grid
  {
      temp[lindex - RADIUS] = in[gindex - RADIUS];
      temp[lindex + BLOCKSIZE] = 0;
  }

  else
  {
    temp[lindex - RADIUS] = in[gindex - RADIUS];
    temp[lindex + BLOCKSIZE] = in[gindex + BLOCKSIZE];
  }

  // Apply the stencil
  int result = 0;
  for (int offset = -RADIUS; offset <= RADIUS; offset++)
  {
    if ( lindex + offset < dim && lindex + offset > -1)
	       result += temp[lindex + offset];
  }

    // Store the result
  out[gindex] = result;

  __syncthreads();

}

#define True  1
#define False 0
void checkResults(int *h_in, int *h_out, int DoCheck=True) {
   // DO NOT CHANGE THIS CODE.
   // CPU calculates the stencil from data in *h_in
   // if DoCheck is True (default) it compares it with *h_out
   // to check the operation of this code.
   // If DoCheck is set to False, it can be used to time the CPU.
   int i, j, ij, result, err;
   err = 0;
   for (i=0; i<N; i++){  // major index.
      result = 0;
      for (j=-RADIUS; j<=RADIUS; j++){
         ij = i+j;
         if (ij>=0 && ij<N)
            result += h_in[ij];
      }
      if (DoCheck) {  // print out some errors for debugging purposes.
         if (h_out[i] != result) { // count errors.
            err++;
            if (err < 8) { // help debug
               printf("h_out[%d]=%d should be %d\n",i,h_out[i], result);
            };
         }
      } else {  // for timing purposes.
         h_out[i] = result;
      }
   }

   if (DoCheck) { // report results.
      if (err != 0){
         printf("Error, %d elements do not match!\n", err);
      } else {
         printf("Success! All elements match CPU result.\n");
      }
   }
}


// ------------
// main program
// ------------
int main(void){

  int *h_in, *h_out;
  int *d_in, *d_out;
  long size = N * sizeof(int);
  int i;
  hipEvent_t start, stop;

  // allocate host memory
  h_in = new int[N];
  h_out = new int[N];

  getDeviceProperties();

  // initialize vector
  for (i=0; i<N; i++){
    //    h_in[i] = i+1;
    h_in[i] = 1;
  }

  // allocate device memory
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  cudaErrorCheck();

  // copy input data to device
  hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
  cudaErrorCheck();

  // Apply stencil by launching a sufficient number of blocks
  printf("\n---------------------------\n");
  printf("Launching 1D stencil kernel\n");
  printf("---------------------------\n");
  printf("Vector length     = %ld (%ld MB)\n",N,N*4/1024/1024);
  printf("Stencil radius    = %d\n",RADIUS);

  //----------------------------------------------------------
  // CODE TO RUN AND TIME THE STENCIL KERNEL.
  //----------------------------------------------------------
  newline();
  printThreadSizes();
  start_timer(&start);
  stencil_1D_block<<<gridSize,blockSize>>>(d_in, d_out, N);
  std::cout << "Elapsed time: " << stop_timer(&start, &stop) << " ms" << std::endl;
  // copy results back to host
  hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
  cudaErrorCheck();
  checkResults(h_in, h_out);
  //----------------------------------------------------------

  // deallocate device memory
  hipFree(d_in);
  hipFree(d_out);
  cudaErrorCheck();
  //=====================================================
  // Evaluate total time of execution with just the CPU.
  //=====================================================
  newline();
  std::cout << "Running stencil with the CPU.\n";
  start_timer(&start);
  // Use checkResults to time CPU version of the stencil with False flag.
  checkResults(h_in, h_out, False);
  std::cout << "Elapsed time: " << stop_timer(&start, &stop) << " ms"<< std::endl;
  //=====================================================

  // deallocate host memory
  free(h_in);
  free(h_out);

  return 0;
}
